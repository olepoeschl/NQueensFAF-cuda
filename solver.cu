#include "hip/hip_runtime.h"
#include "solver.cuh"

#include <iostream>
#include <fstream>
#include <string.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>

/*
 * SolverConfig implementation
*/
bool SolverConfig::validate() const {
	return (updateInterval > 0) && (autoSavePercentageStep > 0 && autoSavePercentageStep <= 100) && (autoSavePath.length() > 0);
}

void SolverConfig::readFrom(SolverConfig config) {
	updateInterval = config.updateInterval;
	autoSaveEnabled = config.autoSaveEnabled;
	autoDeleteEnabled = config.autoDeleteEnabled;
	autoSavePercentageStep = config.autoSavePercentageStep;
	autoSavePath = config.autoSavePath;
}

void SolverConfig::readFrom(std::ifstream in) {

}

void SolverConfig::writeTo(std::ofstream out) const {

}

/*
 * Solver implementation
*/
void Solver::solveAsync() {
	m_solverThread = new std::thread(&Solver::solve, this);
}

void Solver::waitFor() {
	if (m_solverThread == NULL)
		throw SolverException("Solver is not running and therefore cannot be waited for!");
	m_solverThread->join();
	delete m_solverThread;
}

/*
 * Constellation implementation
*/
cuda_constellation Constellation::toCUDAConstellation() {
	cuda_constellation cu;
	cu.ld = ld;
	cu.rd = rd;
	cu.col = col;
	cu.start_ijkl = startIjkl;
	return cu;
}

/*
 * ConstellationsGenerator implementation
*/
ConstellationsGenerator::ConstellationsGenerator(uint8_t N) :
	m_N(N), m_preQueens(0), m_LD(0), m_RD(0), m_subconstellationsCounter(0) {

	m_L = (1 << (m_N - 1));
	m_mask = (m_L << 1) - 1;
}

std::vector<Constellation>& ConstellationsGenerator::genConstellations(uint8_t preQueens) {
	m_preQueens = preQueens;
	uint32_t ld, rd, col, ijkl;
	size_t currentSize;
	const int halfN = (m_N + 1) / 2;

	// calculating start constellations with one Queen on the corner square
	// (N-1,N-1)
	for (uint8_t k = 1; k < m_N - 2; k++) { // j is idx of Queen in last row
		for (uint8_t i = k + 1; i < m_N - 1; i++) { // l is idx of Queen in last col
			// always add the constellation, we can not accidently get symmetric ones
			m_ijkls.insert(toIjkl(i, m_N - 1, k, m_N - 1));

			// occupation of ld, rd according to row 1
			// queens i and k
			ld = (m_L >> (k - 1)) | (m_L >> (i - 1));
			// queens i and l
			rd = (m_L >> (i + 1)) | (m_L >> 1);
			// left border from k, right border from l, also bits i and j from the
			// corresponding
			// queens
			col = 1 | m_L | (m_L >> i);

			// diagonals, that are occupied in the last row by the queen j or l
			// we are going to shift them upwards the board later
			// from queen j and l (same, since queen is in the corner)
			m_LD = 1;
			// from queen k and l
			m_RD = 1 | (1 << k);

			// counter of subconstellations, that arise from setting extra queens
			m_subconstellationsCounter = 0;

			// generate all subconstellations with 5 queens
			setPreQueens(ld, rd, col, k, 0, 1, 3);
			// jam j and k and l together into one integer
			ijkl = toIjkl(i, m_N - 1, k, m_N - 1);

			currentSize = m_constellations.size();

			// ijkl and sym are the same for all subconstellations
			for (uint32_t a = 0; a < m_subconstellationsCounter; a++) {
				uint32_t start = m_constellations.at(currentSize - a - 1).startIjkl;
				m_constellations.at(currentSize - a - 1).startIjkl = start | ijkl;
			}
		}
	}
	// calculate starting constellations for no Queens in corners
	// have a look in the loop above for missing explanations
	for (uint8_t j = 1; j < halfN; j++) { // go through last row
		for (uint8_t l = j + 1; l < m_N - 1; l++) { // go through last col
			for (uint8_t k = m_N - j - 2; k > 0; k--) { // go through first col
				if (k == l) // skip if occupied
					continue;
				for (uint8_t i = j + 1; i < m_N - 1; i++) { // go through first row
					if (i == m_N - 1 - l || i == k) // skip if occupied
						continue;
					// check, if we already found a symmetric constellation
					if (!checkRotations(i, j, k, l)) {
						m_ijkls.insert(toIjkl(i, j, k, l));

						// occupy the board corresponding to the queens on the borders of the
						// board
						ld = (m_L >> (i - 1)) | (1 << (m_N - k));
						rd = (m_L >> (i + 1)) | (1 << (l - 1));
						col = 1 | m_L | (m_L >> j) | (m_L >> i);
						// occupy diagonals of the queens j k l in the last row
						// later we are going to shift them upwards the board
						m_LD = (m_L >> j) | (m_L >> l);
						m_RD = (m_L >> j) | (1 << k);

						// counts all subconstellations
						m_subconstellationsCounter = 0;
						// generate all subconstellations
						setPreQueens(ld, rd, col, k, l, 1, 4);
						// jam j and k and l into one integer
						ijkl = toIjkl(i, j, k, l);

						currentSize = m_constellations.size();

						// jkl and sym and start are the same for all subconstellations
						for (uint32_t a = 0; a < m_subconstellationsCounter; a++) {
							uint32_t start = m_constellations.at(currentSize - a - 1).startIjkl;
							m_constellations.at(currentSize - a - 1).startIjkl = start | ijkl;
						}
					}
				}
			}
		}
	}
	return m_constellations;
}

void ConstellationsGenerator::setPreQueens(uint32_t ld, uint32_t rd, uint32_t col, uint8_t k, uint8_t l, uint8_t row, uint8_t queens) {
	// in row k and l just go further
	if (row == k || row == l) {
		setPreQueens(ld << 1, rd >> 1, col, k, l, row + 1, queens);
		return;
	}
	// add queens until we have preQueens queens
	if (queens == m_preQueens) {
		// add the subconstellations to the list
		// id=-1 signals that this constellation does not have a specific id
		// solutions=-1 signals that this constellation has not been solved yet
		Constellation c(-1, ld, rd, col, row << 20, -1);
		m_constellations.push_back(c);
		m_subconstellationsCounter++;
		return;
	}
	// if not done or row k or l, just place queens and occupy the board and go
	// further
	else {
		int free = ~(ld | rd | col | (m_LD >> (m_N - 1 - row)) | (m_RD << (m_N - 1 - row))) & m_mask;
		int bit;

		while (free > 0) {
			bit = free & (-free);
			free -= bit;
			setPreQueens((ld | bit) << 1, (rd | bit) >> 1, col | bit, k, l, row + 1, queens + 1);
		}
	}
}

uint32_t ConstellationsGenerator::toIjkl(uint8_t i, uint8_t j, uint8_t k, uint8_t l) const {
	return (i << 15) + (j << 10) + (k << 5) + l;
}

bool ConstellationsGenerator::checkRotations(uint8_t i, uint8_t j, uint8_t k, uint8_t l) const {
	// rot90
	if (m_ijkls.count(((m_N - 1 - k) << 15) + ((m_N - 1 - l) << 10) + (j << 5) + i))
		return true;

	// rot180
	if (m_ijkls.count(((m_N - 1 - j) << 15) + ((m_N - 1 - i) << 10) + ((m_N - 1 - l) << 5) + m_N - 1 - k))
		return true;

	// rot270
	if (m_ijkls.count((l << 15) + (k << 10) + ((m_N - 1 - i) << 5) + m_N - 1 - j))
		return true;

	return false;
}

void ConstellationsGenerator::addPseudoConstellation(std::vector<Constellation>& constellations, uint32_t ijkl) {
	constellations.push_back(Constellation(-1, (1 << m_N) - 1, (1 << m_N) - 1, (1 << m_N) - 1, (69 << 20) | ijkl, -2));
}

std::vector<Constellation> ConstellationsGenerator::fillWithPseudoConstellations(std::vector<Constellation>& constellations, uint16_t blockSize) {
	std::sort(constellations.begin(), constellations.end());
	std::vector<Constellation> newConstellations;
	int currentJkl = constellations.at(0).startIjkl & ((1 << 15) - 1);
	for (const auto& c : constellations) {
		// iterate through constellations, add each remaining constellations and fill up
		// each group of ijkl till its dividable by workgroup-size
		if (c.solutions >= 0)
			continue;

		if ((c.startIjkl & ((1 << 15) - 1)) != currentJkl) { // check if new ijkl is found
			while (newConstellations.size() % blockSize != 0) {
				addPseudoConstellation(newConstellations, currentJkl);
			}
			currentJkl = c.startIjkl & ((1 << 15) - 1);
		}
		newConstellations.push_back(c);
	}
	while (newConstellations.size() % blockSize != 0) {
		addPseudoConstellation(newConstellations, currentJkl);
	}
	return newConstellations;
}

/*
 * CUDASolver::Device implementation
*/
std::vector<CUDASolver::Device> CUDASolver::m_availableDevices;

void CUDASolver::Device::createCUDAObjects() {
	checkCUErr(hipCtxCreate(&context, 0, device));
	checkCUErr(hipStreamCreateWithFlags(&xStream, 0));
	checkCUErr(hipStreamCreateWithFlags(&memStream, 0));
	checkCUErr(hipStreamCreateWithFlags(&updateStream, 0));
	checkCUErr(hipEventCreateWithFlags(&startEvent, 0));
	checkCUErr(hipEventCreateWithFlags(&endEvent, 0));
	checkCUErr(hipEventCreateWithFlags(&memEvent, 0));
	checkCUErr(hipEventCreateWithFlags(&updateEvent, 0));
}

void CUDASolver::Device::destroyCUDAObjects() {
	checkCUErr(hipEventDestroy(startEvent));
	checkCUErr(hipEventDestroy(endEvent));
	checkCUErr(hipEventDestroy(memEvent));
	checkCUErr(hipEventDestroy(updateEvent));
	checkCUErr(hipStreamDestroy(xStream));
	checkCUErr(hipStreamDestroy(memStream));
	checkCUErr(hipStreamDestroy(updateStream));
	checkCUErr(hipCtxDestroy(context));
}

/*
 * CUDASolver implementation
*/
CUDASolver::CUDASolver(uint8_t N) : Solver(N) {
	static bool initialized;
	if (!initialized) {
		initialized = true;
		checkCUErr(hipInit(0));
		fetchAvailableDevices();
	}
}

void CUDASolver::checkCUErr(hipError_t err) {
	if (err != hipSuccess) {
		if(err == hipErrorNotInitialized)
			throw std::runtime_error("CUDA was not initialized");

		const char* name;
		int err2 = hipDrvGetErrorName(err, &name);
		if(err2 == hipErrorInvalidValue)
			throw std::runtime_error("unknown CUDA error code: " + std::to_string(err));

		const char* description;
		hipDrvGetErrorName(err, &description);

		std::string errMsg = name + std::string(": ") + description;
		throw std::runtime_error(std::string("CUDA error: " + errMsg));
	}
}

void CUDASolver::checkNVRTCErr(hiprtcResult err) {
	if (err != HIPRTC_SUCCESS) {
		const char* description = hiprtcGetErrorString(err);
		if (!strcmp(description, "NVRTC_ERROR unknown"))
			throw std::runtime_error("unknown NVRTC error code: " + std::to_string(err));
		throw std::runtime_error(std::string("NVRTC error: " + std::string(description)));
	}
}

void CUDASolver::fetchAvailableDevices() {
	int deviceCount;
	checkCUErr(hipGetDeviceCount(&deviceCount));
	for (int i = 0; i < deviceCount; i++) {
		Device device;
		checkCUErr(hipDeviceGet(&(device.device), i));
		checkCUErr(hipDeviceGetName(device.name, 50, device.device));
		m_availableDevices.push_back(device);
	}
}

std::vector<std::string> CUDASolver::getAvailableDevices() const {
	std::vector<std::string> deviceNames;
	for (const Device& device : m_availableDevices) {
		deviceNames.push_back(device.name);
	}
	return deviceNames;
}

void CUDASolver::setDevice(uint8_t index) {
	if (index >= m_availableDevices.size())
		throw std::invalid_argument("invalid device index");
	m_device = m_availableDevices.at(index);
}

void CUDASolver::setPreQueens(uint8_t preQueens) {
	m_preQueens = preQueens;
}

void CUDASolver::compileProgram(const char* kernelSourcePath) {
	std::ifstream kernelSourceFile(kernelSourcePath);
	std::string kernelSource((std::istreambuf_iterator<char>(kernelSourceFile)), (std::istreambuf_iterator<char>()));
	hiprtcProgram program;
	checkNVRTCErr(hiprtcCreateProgram(&program, &kernelSource[0], "nqfaf", 0, NULL, NULL));

	std::string optN = "-D N=" + std::to_string(m_N);
	std::string optBlockSize = "-D BLOCK_SIZE=" + std::to_string(m_device.config.blockSize);
	const char* const options[]{optN.c_str(), optBlockSize.c_str()};
	hiprtcResult err = hiprtcCompileProgram(program, 2, options);
	if (err != HIPRTC_SUCCESS) {
		if (err == HIPRTC_ERROR_COMPILATION) {
			size_t logSize;
			checkNVRTCErr(hiprtcGetProgramLogSize(program, &logSize));
			if (logSize > 1) {
				char* log = new char[logSize];
				checkNVRTCErr(hiprtcGetProgramLog(program, log));
				throw std::invalid_argument("kernel source code could not be compiled: " + std::string(log));
			}
		}
		else {
			checkNVRTCErr(err);
		}
	}
	
	size_t ptxSize;
	checkNVRTCErr(hiprtcGetCodeSize(program, &ptxSize));
	char* ptx = new char[ptxSize];
	checkNVRTCErr(hiprtcGetCode(program, ptx));
	hipModule_t module;
	checkCUErr(hipModuleLoadData(&module, (const void*)ptx));
	checkCUErr(hipModuleGetFunction(&function, module, "nqfaf"));
}

void CUDASolver::solve() {
	ConstellationsGenerator generator(m_N);
	m_constellations = generator.genConstellations(m_preQueens);
	m_constellations = generator.fillWithPseudoConstellations(m_constellations, m_device.config.blockSize);

	m_device.createCUDAObjects();
	compileProgram("kernel.cu");

	// write buffers to GPU
	size_t constellationsSize = m_constellations.size() * sizeof(cuda_constellation);
	cuda_constellation* h_constellations = (cuda_constellation*)malloc(constellationsSize);
	if (h_constellations == NULL) {
		throw std::runtime_error("could not allocate memory for the host array containing the devices constellations");
	}
	std::vector<Constellation>::iterator it;
	int i;
	for (it = m_constellations.begin(), i = 0; it < m_constellations.end(); it++, i++) {
		h_constellations[i] = (*it).toCUDAConstellation();
	}
	size_t resultsSize = m_constellations.size() * sizeof(uint64_t);
	uint64_t* h_results = (uint64_t*)malloc(resultsSize);
	if (h_results == NULL) {
		throw std::runtime_error("could not allocate memory for the host array containing the results of the constellations");
	}
	for (it = m_constellations.begin(), i = 0; it < m_constellations.end(); it++, i++) {
		h_results[i] = (*it).solutions;
	}

	hipDeviceptr_t d_constellations;
	checkCUErr(hipMalloc(&d_constellations, constellationsSize));
	hipDeviceptr_t d_results;
	checkCUErr(hipMalloc(&d_results, resultsSize));

	checkCUErr(hipMemcpyHtoD(d_constellations, h_constellations, constellationsSize));
	checkCUErr(hipMemcpyHtoD(d_results, h_results, resultsSize));
	// ---

	// launch kernel
	// ...

	checkCUErr(hipFree(d_constellations));
	checkCUErr(hipFree(d_results));
	if (h_constellations)
		free(h_constellations);
	if (h_results)
		free(h_results);
	m_device.destroyCUDAObjects();
}

uint64_t CUDASolver::getDuration() const {
	return 69;
}

float CUDASolver::getProgress() const {
	return 1.234f;
}

int64_t CUDASolver::getSolutions() const {
	return 420;
}
