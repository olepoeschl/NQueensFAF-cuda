#include "solver.cuh"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

/*
 * SolverConfig implementation
*/
bool SolverConfig::validate() {
	return (updateInterval > 0) && (autoSavePercentageStep > 0 && autoSavePercentageStep <= 100) && (autoSavePath.length() > 0);
}

void SolverConfig::readFrom(SolverConfig config) {
	updateInterval = config.updateInterval;
	autoSaveEnabled = config.autoSaveEnabled;
	autoDeleteEnabled = config.autoDeleteEnabled;
	autoSavePercentageStep = config.autoSavePercentageStep;
	autoSavePath = config.autoSavePath;
}

void SolverConfig::readFrom(std::ifstream in) {

}

void SolverConfig::writeTo(std::ofstream out) {

}

/*
 * Solver implementation
*/
void Solver::solveAsync() {
	m_solverThread = new std::thread(&Solver::solve, this);
}

void Solver::waitFor() {
	if (m_solverThread == NULL)
		throw SolverException("Solver is not running and therefore cannot be waited for!");
	m_solverThread->join();
	delete m_solverThread;
}

/*
 * CUDASolver implementation
*/
CUDASolver::CUDASolver() : Solver() {
	fetchAvailableDevices();
}

void CUDASolver::checkCUErr(hipError_t err) {
	if (err != hipSuccess) {
		std::string errMsg = "";
		const char* strBuf = NULL;

		int err2 = hipDrvGetErrorName(err, &strBuf);
		if (err2 == hipErrorInvalidValue) {
			throw std::runtime_error("unknown CUDA error code: " + err2);
		}
		errMsg.append(strBuf);
		const char* strBuf2 = NULL;
		err2 = hipDrvGetErrorString(err, &strBuf2);
		if (err2 == hipErrorInvalidValue) {
			throw std::runtime_error("unknown CUDA error code: " + err2);
		}
		errMsg.append(strBuf2);

		throw std::runtime_error(std::string("CUDA error: " + errMsg));
	}
}

void CUDASolver::fetchAvailableDevices() {
	int deviceCount;
	checkCUErr(hipGetDeviceCount(&deviceCount));
	std::cout << "device count: " << deviceCount << std::endl;
}

void CUDASolver::solve() {
	hipInit(0);
	std::cout << "solving..." << std::endl;
}

int64_t CUDASolver::getDuration() {
	return 69;
}

float CUDASolver::getProgress() {
	return 1.234f;
}

int64_t CUDASolver::getSolutions() {
	return 420;
}
