#include "solver.cuh"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

/*
 * SolverConfig implementation
*/
bool SolverConfig::validate() const {
	return (updateInterval > 0) && (autoSavePercentageStep > 0 && autoSavePercentageStep <= 100) && (autoSavePath.length() > 0);
}

void SolverConfig::readFrom(SolverConfig config) {
	updateInterval = config.updateInterval;
	autoSaveEnabled = config.autoSaveEnabled;
	autoDeleteEnabled = config.autoDeleteEnabled;
	autoSavePercentageStep = config.autoSavePercentageStep;
	autoSavePath = config.autoSavePath;
}

void SolverConfig::readFrom(std::ifstream in) {

}

void SolverConfig::writeTo(std::ofstream out) const {

}

/*
 * Solver implementation
*/
void Solver::solveAsync() {
	m_solverThread = new std::thread(&Solver::solve, this);
}

void Solver::waitFor() {
	if (m_solverThread == NULL)
		throw SolverException("Solver is not running and therefore cannot be waited for!");
	m_solverThread->join();
	delete m_solverThread;
}

/*
 * ConstellationsGenerator implementation
*/
ConstellationsGenerator::ConstellationsGenerator(uint8_t N) :
	m_N(N), m_preQueens(0), m_LD(0), m_RD(0), m_subconstellationsCounter(0) {

	m_L = (1 << (m_N - 1));
	m_mask = (m_L << 1) - 1;
}

std::vector<Constellation>& ConstellationsGenerator::genConstellations(uint8_t preQueens) {
	m_preQueens = preQueens;
	uint32_t ld, rd, col, ijkl;
	size_t currentSize;
	const int halfN = (m_N + 1) / 2;

	// calculating start constellations with one Queen on the corner square
	// (N-1,N-1)
	for (uint8_t k = 1; k < m_N - 2; k++) { // j is idx of Queen in last row
		for (uint8_t i = k + 1; i < m_N - 1; i++) { // l is idx of Queen in last col
			// always add the constellation, we can not accidently get symmetric ones
			m_ijkls.insert(toIjkl(i, m_N - 1, k, m_N - 1));

			// occupation of ld, rd according to row 1
			// queens i and k
			ld = (m_L >> (k - 1)) | (m_L >> (i - 1));
			// queens i and l
			rd = (m_L >> (i + 1)) | (m_L >> 1);
			// left border from k, right border from l, also bits i and j from the
			// corresponding
			// queens
			col = 1 | m_L | (m_L >> i);

			// diagonals, that are occupied in the last row by the queen j or l
			// we are going to shift them upwards the board later
			// from queen j and l (same, since queen is in the corner)
			m_LD = 1;
			// from queen k and l
			m_RD = 1 | (1 << k);

			// counter of subconstellations, that arise from setting extra queens
			m_subconstellationsCounter = 0;

			// generate all subconstellations with 5 queens
			setPreQueens(ld, rd, col, k, 0, 1, 3);
			// jam j and k and l together into one integer
			ijkl = toIjkl(i, m_N - 1, k, m_N - 1);

			currentSize = m_constellations.size();

			// ijkl and sym are the same for all subconstellations
			for (uint32_t a = 0; a < m_subconstellationsCounter; a++) {
				uint32_t start = m_constellations.at(currentSize - a - 1).startijkl;
				m_constellations.at(currentSize - a - 1).startijkl = start | ijkl;
			}
		}
	}
	// calculate starting constellations for no Queens in corners
	// have a look in the loop above for missing explanations
	for (uint8_t j = 1; j < halfN; j++) { // go through last row
		for (uint8_t l = j + 1; l < m_N - 1; l++) { // go through last col
			for (uint8_t k = m_N - j - 2; k > 0; k--) { // go through first col
				if (k == l) // skip if occupied
					continue;
				for (uint8_t i = j + 1; i < m_N - 1; i++) { // go through first row
					if (i == m_N - 1 - l || i == k) // skip if occupied
						continue;
					// check, if we already found a symmetric constellation
					if (!checkRotations(i, j, k, l)) {
						m_ijkls.insert(toIjkl(i, j, k, l));

						// occupy the board corresponding to the queens on the borders of the
						// board
						ld = (m_L >> (i - 1)) | (1 << (m_N - k));
						rd = (m_L >> (i + 1)) | (1 << (l - 1));
						col = 1 | m_L | (m_L >> j) | (m_L >> i);
						// occupy diagonals of the queens j k l in the last row
						// later we are going to shift them upwards the board
						m_LD = (m_L >> j) | (m_L >> l);
						m_RD = (m_L >> j) | (1 << k);

						// counts all subconstellations
						m_subconstellationsCounter = 0;
						// generate all subconstellations
						setPreQueens(ld, rd, col, k, l, 1, 4);
						// jam j and k and l into one integer
						ijkl = toIjkl(i, j, k, l);

						currentSize = m_constellations.size();

						// jkl and sym and start are the same for all subconstellations
						for (uint32_t a = 0; a < m_subconstellationsCounter; a++) {
							uint32_t start = m_constellations.at(currentSize - a - 1).startijkl;
							m_constellations.at(currentSize - a - 1).startijkl = start | ijkl;
						}
					}
				}
			}
		}
	}
	return m_constellations;
}

void ConstellationsGenerator::setPreQueens(uint32_t ld, uint32_t rd, uint32_t col, uint8_t k, uint8_t l, uint8_t row, uint8_t queens) {
	// in row k and l just go further
	if (row == k || row == l) {
		setPreQueens(ld << 1, rd >> 1, col, k, l, row + 1, queens);
		return;
	}
	// add queens until we have preQueens queens
	if (queens == m_preQueens) {
		// add the subconstellations to the list
		// TODO: solutions=-1 signals that this constellation has not been solved yet
		Constellation c(0, ld, rd, col, row << 20, -1);
		m_constellations.push_back(c);
		m_subconstellationsCounter++;
		return;
	}
	// if not done or row k or l, just place queens and occupy the board and go
	// further
	else {
		int free = ~(ld | rd | col | (m_LD >> (m_N - 1 - row)) | (m_RD << (m_N - 1 - row))) & m_mask;
		int bit;

		while (free > 0) {
			bit = free & (-free);
			free -= bit;
			setPreQueens((ld | bit) << 1, (rd | bit) >> 1, col | bit, k, l, row + 1, queens + 1);
		}
	}
}

uint32_t ConstellationsGenerator::toIjkl(uint8_t i, uint8_t j, uint8_t k, uint8_t l) const {
	return (i << 15) + (j << 10) + (k << 5) + l;
}

bool ConstellationsGenerator::checkRotations(uint8_t i, uint8_t j, uint8_t k, uint8_t l) const {
	// rot90
	if (m_ijkls.count(((m_N - 1 - k) << 15) + ((m_N - 1 - l) << 10) + (j << 5) + i))
		return true;

	// rot180
	if (m_ijkls.count(((m_N - 1 - j) << 15) + ((m_N - 1 - i) << 10) + ((m_N - 1 - l) << 5) + m_N - 1 - k))
		return true;

	// rot270
	if (m_ijkls.count((l << 15) + (k << 10) + ((m_N - 1 - i) << 5) + m_N - 1 - j))
		return true;

	return false;
}

/*
 * CUDASolver implementation
*/
std::vector<CUDASolver::Device> CUDASolver::m_availableDevices;
CUDASolver::CUDASolver(uint8_t N) : Solver(N) {
	static bool initialized;
	if (!initialized) {
		initialized = true;
		checkCUErr(hipInit(0));
		fetchAvailableDevices();
	}
}

void CUDASolver::checkCUErr(hipError_t err) {
	if (err != hipSuccess) {
		if(err == hipErrorNotInitialized)
			throw std::runtime_error("CUDA was not initialized");

		const char* name;
		int err2 = hipDrvGetErrorName(err, &name);
		if(err2 == hipErrorInvalidValue)
			throw std::runtime_error("unknown CUDA error code: " + std::to_string(err));
		std::cout << "name: " << name << std::endl;

		const char* description;
		hipDrvGetErrorName(err, &description);
		std::cout << "description: " << description << std::endl;

		std::string errMsg = name + std::string(": ") + description;
		throw std::runtime_error(std::string("CUDA error: " + errMsg));
	}
}

void CUDASolver::fetchAvailableDevices() {
	int deviceCount;
	checkCUErr(hipGetDeviceCount(&deviceCount));
	for (int i = 0; i < deviceCount; i++) {
		Device device;
		checkCUErr(hipDeviceGet(&(device.device), i));
		checkCUErr(hipDeviceGetName(device.name, 50, device.device));
		m_availableDevices.push_back(device);
	}
}

std::vector<std::string> CUDASolver::getAvailableDevices() const {
	std::vector<std::string> deviceNames;
	for (const Device& device : m_availableDevices) {
		deviceNames.push_back(device.name);
	}
	return deviceNames;
}

void CUDASolver::setDevice(uint8_t index) {
	if (index >= m_availableDevices.size())
		throw std::invalid_argument("invalid device index");
	m_device = m_availableDevices.at(index);
}

void CUDASolver::solve() {
	ConstellationsGenerator generator(m_N);
	m_constellations = generator.genConstellations(6);
	std::cout << "generated " << m_constellations.size() << " constellations" << std::endl;
}

uint64_t CUDASolver::getDuration() const {
	return 69;
}

float CUDASolver::getProgress() const {
	return 1.234f;
}

int64_t CUDASolver::getSolutions() const {
	return 420;
}
