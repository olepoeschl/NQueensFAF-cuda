#include "solver.cuh"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

/*
 * SolverConfig implementation
*/
bool SolverConfig::validate() {
	return (updateInterval > 0) && (autoSavePercentageStep > 0 && autoSavePercentageStep <= 100) && (autoSavePath.length() > 0);
}

void SolverConfig::readFrom(SolverConfig config) {
	updateInterval = config.updateInterval;
	autoSaveEnabled = config.autoSaveEnabled;
	autoDeleteEnabled = config.autoDeleteEnabled;
	autoSavePercentageStep = config.autoSavePercentageStep;
	autoSavePath = config.autoSavePath;
}

void SolverConfig::readFrom(std::ifstream in) {

}

void SolverConfig::writeTo(std::ofstream out) {

}

/*
 * Solver implementation
*/
void Solver::solveAsync() {
	m_solverThread = new std::thread(&Solver::solve, this);
}

void Solver::waitFor() {
	if (m_solverThread == NULL)
		throw SolverException("Solver is not running and therefore cannot be waited for!");
	m_solverThread->join();
	delete m_solverThread;
}

/*
 * CUDASolver implementation
*/
CUDASolver::CUDASolver() : Solver() {
	static bool initialized;
	if (!initialized) {
		initialized = true;
		checkCUErr(hipInit(0));
	}
	fetchAvailableDevices();
}

void CUDASolver::checkCUErr(hipError_t err) {
	if (err != hipSuccess) {
		if(err == hipErrorNotInitialized)
			throw std::runtime_error("CUDA was not initialized");

		const char* name;
		int err2 = hipDrvGetErrorName(err, &name);
		if(err2 == hipErrorInvalidValue)
			throw std::runtime_error("unknown CUDA error code: " + std::to_string(err));
		std::cout << "name: " << name << std::endl;

		const char* description;
		hipDrvGetErrorName(err, &description);
		std::cout << "description: " << description << std::endl;

		std::string errMsg = name + std::string(": ") + description;
		throw std::runtime_error(std::string("CUDA error: " + errMsg));
	}
}

void CUDASolver::fetchAvailableDevices() {
	int deviceCount;
	checkCUErr(hipGetDeviceCount(&deviceCount));
	for (int i = 0; i < deviceCount; i++) {
		Device device;
		checkCUErr(hipDeviceGet(&(device.device), i));
		checkCUErr(hipDeviceGetName(device.name, 50, device.device));
		m_availableDevices.push_back(device);
	}
}

std::vector<std::string> CUDASolver::getAvailableDevices() {
	std::vector<std::string> deviceNames;
	for (const Device& device : m_availableDevices) {
		deviceNames.push_back(device.name);
	}
	return deviceNames;
}

void CUDASolver::setDevice(uint8_t index) {
	if (index >= m_availableDevices.size())
		throw std::invalid_argument("invalid device index");
	m_device = m_availableDevices.at(index);
}

void CUDASolver::solve() {
	std::cout << "solving..." << std::endl;
}

int64_t CUDASolver::getDuration() {
	return 69;
}

float CUDASolver::getProgress() {
	return 1.234f;
}

int64_t CUDASolver::getSolutions() {
	return 420;
}
