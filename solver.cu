#include "solver.cuh"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

/*
 * SolverConfig implementation
*/
bool SolverConfig::validate() const {
	return (updateInterval > 0) && (autoSavePercentageStep > 0 && autoSavePercentageStep <= 100) && (autoSavePath.length() > 0);
}

void SolverConfig::readFrom(SolverConfig config) {
	updateInterval = config.updateInterval;
	autoSaveEnabled = config.autoSaveEnabled;
	autoDeleteEnabled = config.autoDeleteEnabled;
	autoSavePercentageStep = config.autoSavePercentageStep;
	autoSavePath = config.autoSavePath;
}

void SolverConfig::readFrom(std::ifstream in) {

}

void SolverConfig::writeTo(std::ofstream out) const {

}

/*
 * Solver implementation
*/
void Solver::solveAsync() {
	m_solverThread = new std::thread(&Solver::solve, this);
}

void Solver::waitFor() {
	if (m_solverThread == NULL)
		throw SolverException("Solver is not running and therefore cannot be waited for!");
	m_solverThread->join();
	delete m_solverThread;
}

/*
 * ConstellationsGenerator implementation
*/
ConstellationsGenerator::ConstellationsGenerator(int N) :
	m_N(N), m_preQueens(0), m_LD(0), m_RD(0), m_subconstellationsCounter(0) {

	m_L = (1 << (m_N - 1));
	m_mask = (m_L << 1) - 1;
}

/*
 * CUDASolver implementation
*/
std::vector<CUDASolver::Device> CUDASolver::m_availableDevices;
CUDASolver::CUDASolver() : Solver() {
	static bool initialized;
	if (!initialized) {
		initialized = true;
		checkCUErr(hipInit(0));
		fetchAvailableDevices();
	}
}

void CUDASolver::checkCUErr(hipError_t err) {
	if (err != hipSuccess) {
		if(err == hipErrorNotInitialized)
			throw std::runtime_error("CUDA was not initialized");

		const char* name;
		int err2 = hipDrvGetErrorName(err, &name);
		if(err2 == hipErrorInvalidValue)
			throw std::runtime_error("unknown CUDA error code: " + std::to_string(err));
		std::cout << "name: " << name << std::endl;

		const char* description;
		hipDrvGetErrorName(err, &description);
		std::cout << "description: " << description << std::endl;

		std::string errMsg = name + std::string(": ") + description;
		throw std::runtime_error(std::string("CUDA error: " + errMsg));
	}
}

void CUDASolver::fetchAvailableDevices() {
	int deviceCount;
	checkCUErr(hipGetDeviceCount(&deviceCount));
	for (int i = 0; i < deviceCount; i++) {
		Device device;
		checkCUErr(hipDeviceGet(&(device.device), i));
		checkCUErr(hipDeviceGetName(device.name, 50, device.device));
		m_availableDevices.push_back(device);
	}
}

std::vector<std::string> CUDASolver::getAvailableDevices() const {
	std::vector<std::string> deviceNames;
	for (const Device& device : m_availableDevices) {
		deviceNames.push_back(device.name);
	}
	return deviceNames;
}

void CUDASolver::setDevice(uint8_t index) {
	if (index >= m_availableDevices.size())
		throw std::invalid_argument("invalid device index");
	m_device = m_availableDevices.at(index);
}

void CUDASolver::solve() {
	std::cout << "solving..." << std::endl;
}

int64_t CUDASolver::getDuration() const {
	return 69;
}

float CUDASolver::getProgress() const {
	return 1.234f;
}

int64_t CUDASolver::getSolutions() const {
	return 420;
}
